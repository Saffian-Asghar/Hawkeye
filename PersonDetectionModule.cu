#include "hip/hip_runtime.h"
#include "PersonDetectionModule.cuh"
#include <fstream>
#include<time.h>
#include <stdlib.h>

__global__ void cuda_masking(unsigned char* cimg, unsigned char* modelImg, unsigned char* modelImg1, unsigned char* retimg, int rows, int cols, int channels, int blvl) {
	int x = getIndex(blockIdx.x, blockDim.x, threadIdx.x);
	int y = getIndex(blockIdx.y, blockDim.y, threadIdx.y);
	if (x >= rows || y >= cols || channels < 3)
		return;
	int point = pixelI(x, y, cols, 1);  // index of a pixel
	int index = point * channels;  // index of a pixel

	// grey
	double grey = get_grey(cimg[index + 0], cimg[index + 1], cimg[index + 2]); // pixel value


	// model update

	modelImg[point] = (unsigned char)(modelImg[point] * 0.85 + grey * 0.15);

	// diff
	double diff = abs((int)modelImg1[point] - grey);
	
	modelImg1[point] = (unsigned char)(modelImg1[point] * 0.15 + modelImg[point] * 0.85);

	//threshould 

	// balck and white
	retimg[point] = (unsigned char)(diff < blvl) ? 0 : 255;
	// grey scale
	retimg[point] = (unsigned char)(diff < blvl) ? 0 : grey;
}

PersonDetection::PersonDetection(int cid) : _blvl(25),camId(cid), rows(0), cols(0), channels(0) {//large
//PersonDetection::PersonDetection(int cid) : _blvl(18),camId(cid), rows(0), cols(0), channels(0) {//small
	cudaStatus = hipSetDevice(0);
	img_no = 0;
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}
}
bool PersonDetection::shouldEnd() {
	return cudaStatus != hipSuccess; 
}
bool PersonDetection::prepare(const Mat& img) {
	rows = img.rows;
	cols = img.cols;
	channels = img.channels();

	_1d_img = rows * cols * sizeof(unsigned char);
	_3d_img = _1d_img * channels;


	cudaStatus = hipMalloc<unsigned char>(&_c_arr, _3d_img);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "prepare() hipMalloc() _c_arr failed!");
		return false;
	}
	cudaStatus = hipMalloc<unsigned char>(&_m_arr, _1d_img);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "prepare() hipMalloc() _m_arr failed!");
		return false;
	}
	cudaStatus = hipMalloc<unsigned char>(&_m_arr1, _1d_img);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "prepare() hipMalloc() _m_arr1 failed!");
		return false;
	}
	cudaStatus = hipMalloc<unsigned char>(&_r_arr, _1d_img);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "prepare() hipMalloc() _r_arr failed!");
		return false;
	}

	_r_frame = Mat::zeros(img.size(), 0);

	cudaStatus = hipMemcpy(_m_arr, _r_frame.ptr(), _1d_img, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "prepare() hipMemcpy _m_arr hipMemcpyHostToDevice failed!");
		return false;
	}cudaStatus = hipMemcpy(_m_arr1, _r_frame.ptr(), _1d_img, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "prepare() hipMemcpy _m_arr1 hipMemcpyHostToDevice failed!");
		return false;
	}
	cudaStatus = hipMemcpy(_r_arr, _r_frame.ptr(), _1d_img, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "prepare() hipMemcpy _r_arr hipMemcpyHostToDevice failed!");
		return false;
	}
	return true;
}
bool PersonDetection::learning(const Mat&img) {
	_c_frame = img.clone();
	cudaStatus = hipMemcpy(_c_arr, img.ptr(), _3d_img, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "learning() hipMemcpy _c_arr hipMemcpyHostToDevice failed!");
		return false;
	}
	dim3 blockDim(1, 1);
	dim3 gridDim(rows, cols);

	cuda_masking<<<gridDim, blockDim >>>(_c_arr, _m_arr, _m_arr1, _r_arr, rows, cols, channels, _blvl);

	cudaStatus = hipMemcpy(_r_frame.data, _r_arr, _1d_img, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "learning() hipMemcpy _r_frame hipMemcpyHostToDevice failed!");
		return false;
	}

	return true;
}

//void PersonDetection::deleteUnwantedPerson() {
//	for (auto& pair : persons) {
//		if (pair.second.second == false)
//			persons.erase(pair.first);
//	}
//	for (auto& pair : persons) {
//		pair.second.second = false;
//	}
//}

void PersonDetection::extraction(map<int, map<uchar, int>> &greyColorVals, bool saveImg){
	
	// _r_frame is segmented/ foreground frame
	//imshow("mask", _r_frame);
	// waiting for background learning


	vector<std::vector<cv::Point>> contours;
	basics.findContours(_r_frame, contours);

	//iou slow
	cv::Rect boundingRectB;
	Rect interect, union_;
	double iou = 0.f;
	//double coutourArea = 200;//small
	double coutourArea = 20000;//large

	// Process each contour to detect and track persons
	for (const auto& contour : contours) {
		double area = cv::contourArea(contour);
		cv::Rect boundingRect = cv::boundingRect(contour);

		if (area > coutourArea) {
			
			Person person = getPerson(_c_frame, _r_frame, boundingRect, greyColorVals);
			Mat personImage = _c_frame(person.boundingRect).clone();
			
			save(personImage, person.id);
			log(person, true, person.boundingRect);

			// Draw bounding box with person ID on the frame
			cv::rectangle(_c_frame, person.boundingRect, cv::Scalar(0, 255, 0), 2);
			cv::putText(_c_frame, std::to_string(person.id), cv::Point(person.boundingRect.x, person.boundingRect.y - 10), cv::FONT_HERSHEY_SIMPLEX, 0.9, cv::Scalar(0, 255, 0), 2);

		}
	}
	//deleteUnwantedPerson();
}

Mat PersonDetection::getOrignalFrame() { return _c_frame; }
Mat PersonDetection::getMaskFrame() { return _r_frame; }

void PersonDetection::printloc(Location l, bool saveImg, Rect boundingRect) {
	char* time = strtok(ctime(&(l.time)), "\n");

	fprintf(stderr, "CamId: %d, PId: %d, Time: %s, loc.x: %f,  loc.y %f \n", l.camId, l.pid, time, l.locX, l.locY);

	string filePath = "objects\\" + to_string(l.pid) + "\\";

	system(("if not exist \"" + filePath + "\" mkdir " + filePath).c_str());

	fstream objectFile;
	objectFile.open(filePath + "path.csv", ios::out | ios::app);
	objectFile << l.camId << "," << l.pid << "," << time << "," << l.locX << "," << l.locX << "\n";
	objectFile.close();

	if (saveImg) {
		Mat img;
		resize(_c_frame(boundingRect), img, Size(200, 200), 0, 0);
		imwrite(filePath + "a.jpg", img);
	}
}

void PersonDetection::save(const Mat& personImage, int id) {
	// Placeholder function to save person image and ID to a database
	// Implement the logic to save the image and ID to your specific database
	if (_show) {
		string str = "ID: ";
		str += (id);
		cv::imshow(str, personImage);
	}
}
void PersonDetection::log(Person p, bool saveImg, Rect boundingRect) {
	Location loc;
	loc.pid = p.id;
	loc.camId = camId;
	loc.locX = p.centroid.x;
	loc.locY = p.centroid.y;
	loc.time = time(NULL);
	printloc(loc, saveImg, boundingRect);
	string str = "x:";
	str += loc.locX;
	str += " y: ";
	str += loc.locY;
	str += " cam: ";
	str += loc.camId;
	str += " pid";
	str += loc.pid;

	//int a = rand();
	//db.updateData(loc);
}
void PersonDetection::showDetectedPerson(bool _s) {
	_show = _s;
}

void PersonDetection::finilize() {
	if (shouldEnd())
		cout << hipGetErrorString(cudaStatus) << endl;

	cudaStatus = hipDeviceReset();

}

float PersonDetection::calculateCentroidDistance(const cv::Point2f& centroid1, const cv::Point2f& centroid2) {
	float x = (centroid1.x - centroid2.x);
	float y = (centroid1.y - centroid2.y);

	return sqrt(pow(x, 2.0f) + pow(y, 2.0f));
}

// Function to compare two blocks using Euclidean distance
double PersonDetection::compareBlocks(vector<double> block1Features, vector<double> block2Features) {
	double distance = 0.0;
	for (size_t i = 0; i < block1Features.size(); i++) {
		double diff = block1Features[i] - block2Features[i];
		distance += diff * diff;
	}
	return sqrt(distance);
}

std::map<uchar, int> getColorsCount(cv::Mat dimage) {
	cv::resize(dimage, dimage, cv::Size(80, 80));
	uchar temp = -1;
	//dct(dimage, dimage);
	std::map<uchar, int> chars;
	for (int i = 0; i < dimage.rows; i++)
		for (int j = 0; j < dimage.cols; j++) {
			temp = dimage.at<uchar>(i, j);
			if (temp != 0 && chars.find(temp) != chars.end())
				chars[temp]++;
			else
				chars.insert(std::pair<uchar, int>(temp, 1));
		}
	return chars;
}



bool PersonDetection::percentCompare(float n, float d) {
	if ((n / d) * 100.f > 60.f)
		return true;
	else
		return false;
}

int PersonDetection::compare(map<uchar, int> currGreyColorVals, map<int, map<uchar, int>> greyColorVals) {
	bestMatchValue = 0;
	matchId = -1;
	for (auto i : greyColorVals) {
		total = pos = percent = 0;

		for (auto e : currGreyColorVals) {
			total++;
			if (e.second > i.second[e.first])
				if (percentCompare(i.second[e.first], e.second))
					pos++;
				else if (percentCompare(e.second, i.second[e.first]))
					pos++;
		}

		percent = (pos / total) * 100.f;
		if (percent > 60 && percent > bestMatchValue) {
			bestMatchValue = percent;
			matchId = i.first;
		}
	}

	return matchId;
}

cv::Point2f PersonDetection::centroidWeightedSum(const std::deque<cv::Point2f>& values, size_t index = 0) {

	if (index + 1 == values.size())
		return values.at(index);
	else
		return 0.2 * values.at(index) + 0.8 * centroidWeightedSum(values, index + 1);
}

void PersonDetection::calculateCentrod(cv::Point2f savedCentrod, const cv::Point2f newCentroid, Person& person, bool calcDiff = false) {

	if (calcDiff) {
		person.positions.push_back(newCentroid - savedCentrod);

		if (person.positions.size() == 7) {
			person.positions.pop_front();
			person.centroid = savedCentrod + centroidWeightedSum(person.positions);
		} else
			person.centroid = newCentroid;
	} else
		person.centroid = newCentroid;
}

Person PersonDetection::getPerson(Mat& frame, Mat& gFrame, cv::Rect boundingRect, map<int, map<uchar, int>>& greyColorVals) {

	Person person, tempPerson;
	person.boundingRect = boundingRect;
	cv::Point2f objCentrod = (boundingRect.tl() + boundingRect.br()) * 0.5f;
	
	float distance = -1, smallestDistance = FLT_MAX;
	int smallestDistanceId = -1;
	for (auto& pair : persons) {
		tempPerson = pair.second.first;
		// Calculate distance between centroids
		distance = calculateCentroidDistance(tempPerson.centroid, objCentrod);
		if (distance < 1000 && distance < smallestDistance) { //large
		//if (distance < 30 && distance < smallestDistance) { //small
			smallestDistance = distance;
			smallestDistanceId = tempPerson.id; 
		}
	}

	if (smallestDistanceId > -1) {
		person.id = smallestDistanceId;
		person.positions = persons[smallestDistanceId].first.positions;
		calculateCentrod(persons[smallestDistanceId].first.centroid, objCentrod, person, true);
		persons[smallestDistanceId] = make_pair(person, true);
		Basics::convertGrey(_r_frame, _p_frame);
		return person;
	}

	map<uchar, int> currGreyColorVals = getColorsCount(gFrame(boundingRect));

	int matchValue = -1;
	if (!  greyColorVals.empty()) {
		matchValue = compare(currGreyColorVals, greyColorVals);
	}

	if (matchValue < 0) {
		cout << "=================================================== 361";
		person.id = personId;
		greyColorVals.insert(pair<int, map<uchar, int>>(personId, currGreyColorVals));
		personId++;
	}
	else {
		cout << "++++++++++++++++++++++++++++++++++++++++++++++++++++++ 367";
		person.id = matchValue;
	}
	calculateCentrod(person.centroid, objCentrod, person);
	persons[person.id].first = person;
	persons[person.id].second = true;

	_p_frame = _r_frame;
	return person;
}

//Person PersonDetection::getPerson(Mat& frame, Mat& gFrame, cv::Rect boundingRect, map<int, map<uchar, int>>& greyColorVals) {
//	cv::Point2f centroid = (boundingRect.tl() + boundingRect.br()) * 0.5f;
//	for (auto& pair : persons) {
//		Person& person = pair.second.first;
//		// Calculate distance between centroids
//		float distance = calculateCentroidDistance(person.centroid, centroid);
//		// Check if the distance is smaller than a threshold
//		cout << distance << endl;
//		if (distance < 1000) { //large
//		//if (distance < 0) { //small
//			cout << "-----------------------------------------------------   : 344 " << endl;
//			person.boundingRect = boundingRect;
//			person.centroid = centroid;
//			persons[personId] = make_pair(person, true);
//			Basics::convertGrey(_r_frame, _p_frame);
//			return person;
//		}
//	}
//
//	map<uchar, int> currGreyColorVals = getColorsCount(gFrame(boundingRect));
//
//	int matchValue = -1;
//	if (!greyColorVals.empty()) {
//		matchValue = compare(currGreyColorVals, greyColorVals);
//	}
//
//	Person Person;
//	if (matchValue < 0) {
//		cout << "=================================================== 361";
//		Person.id = personId;
//		persons[personId].first = Person;
//		persons[personId].second = true;
//		greyColorVals.insert(pair<int, map<uchar, int>>(personId, currGreyColorVals));
//		personId++;
//	}
//	else {
//		cout << "++++++++++++++++++++++++++++++++++++++++++++++++++++++ 367";
//		Person.id = matchValue;
//		persons[matchValue].first.centroid = centroid;
//		persons[matchValue].second = true;
//	}
//
//	Person.boundingRect = boundingRect;
//	Person.centroid = centroid;
//
//	_p_frame = _r_frame;
//	return Person;
//}