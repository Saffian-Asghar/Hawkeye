﻿#pragma once
#include "HawkCam.cuh"
#include <websocketpp/config/asio_no_tls.hpp>
#include <websocketpp/server.hpp>

typedef websocketpp::server<websocketpp::config::asio> server;

void on_open(websocketpp::connection_hdl hdl) {
    connections.insert(hdl);
}

void on_close(websocketpp::connection_hdl hdl) {
    connections.erase(hdl);
}

int main() {
    server s;

    s.set_open_handler(&on_open);
    s.set_close_handler(&on_close);

    s.init_asio();
    s.listen(5892);
    s.start_accept();

    const short int camsCount = 3;
    string paths[camsCount] = { "videos\\v1.mp4",  "videos\\v2.mp4",  "videos\\v3.mp4"};
    HawkCam cams(paths, camsCount);
    cams.loadStreams();
    cams.setupStreams();
    cams.loop();
    cams.finalize();

    s.run();
}