﻿#pragma once
#include "HawkCams.cuh"
#include "crow_all.h"  // Include Crow library

// Global pointer to Crow WebSocket server
crow::SimpleApp* ws_server;

// WebSocket route handler function to broadcast image frames
void handleWebSocketImageFrame(const crow::websocket::connection& conn, const std::string& data) {
    // Broadcast the received image frame to all connected clients
    // Here, you need to implement logic to broadcast 'data' to all connected clients
    // Example: You might send the image data received from 'data' to all connected clients
    // Hint: Use ws_server->broadcast_binary(data) to broadcast data to all clients
}

void working_code() {
    const short int camsCount = 3;
    string paths[camsCount] = { "videos\\v1.mp4",  "videos\\v2.mp4",  "videos\\v3.mp4"};
    HawkCam cams(paths, camsCount);
    cams.loadStreams();
    cams.setupStreams();
    cams.loop();
    cams.finalize();
}

int main() {
    // Start the WebSocket server
    crow::SimpleApp app;
    ws_server = &app;

    // Define WebSocket route to handle image frame messages
    CROW_ROUTE(app, "/image_frame")
        .websocket()
        .onmessage(handleWebSocketImageFrame);

    // Start the server on port 8080
    app.port(8080).multithreaded().run();

    // Start the main functionality of your project
    working_code();

    return 0;
}
