﻿#pragma once
#include "HawkCams.cuh"

void working_code() {
    const short int camsCount = 3;
    string paths[camsCount] = { "videos\\v1.mp4",  "videos\\v2.mp4",  "videos\\v3.mp4"};
    /*const short int camsCount = 1;
    string paths[camsCount] = { "videos\\c4.mp4"};*/
    HawkCam cams(paths, camsCount);
    cams.loadStreams();
    cams.setupStreams();
    cams.loop();
    cams.finalize();
}

int main() {

    working_code();

    return 0;
}